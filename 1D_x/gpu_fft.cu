#include "hip/hip_runtime.h"
/**********************************************************************************
* Numerical Solution for the Cubic Nonlinear Schrodinger Equation        		  *
* using second order split step Fourier method.                                   *
* Coded by: Omar Ashour, Texas A&M University at Qatar, February 2015.    	      *
**********************************************************************************/
#include "../lib/cu_helpers.h"
#include <hipfft/hipfft.h>

// Grid Parameters
#define XN	nodes					// Number of Fourier modes
#define TN	100						// Number of temporal nodes
#define LX	10.0					// x-spatial domain [-LX,LX)
#define TT	10.0            		// Max time
#define DX	(2*LX / XN)				// x-spatial step size
#define DT	(TT / TN)    			// temporal step size

// Timing parameters
#define IRVL  100				// Timing interval. Take a reading every N iterations.

// Output files
#define PLOT_F "gpu_fft_plot.m"
#define TIME_F argv[2]

// Function prototypes
__global__ void nonlin(hipfftDoubleComplex *psi, double dt, int xn);
__global__ void lin(hipfftDoubleComplex *psi, double *k2, double dt, int xn);
__global__ void normalize(hipfftDoubleComplex *psi, int size);

int main(int argc, char *argv[])
{                                                                          
    // Timing info
	hipEvent_t begin_event, end_event;
	hipEventCreate(&begin_event);
	hipEventCreate(&end_event);
    
	// Print basic info about simulation
	const int nodes = atoi(argv[1]);
	printf("XN: %d. DX: %f, DT: %f, dt/dx^2: %f\n", XN, DX, DT, DT/(DX*DX));
	
	// Allocate host arrays
    double *h_x = (double*)malloc(sizeof(double) * XN);
	double *h_k2 = (double*)malloc(sizeof(double) * XN);
	double *h_kx = (double*)malloc(XN * sizeof(double));
	hipfftDoubleComplex *h_psi = (hipfftDoubleComplex*)
										malloc(sizeof(hipfftDoubleComplex)*XN);
	hipfftDoubleComplex *h_psi_0 = (hipfftDoubleComplex*)
										malloc(sizeof(hipfftDoubleComplex)*XN);
	
	// Create transform plans
    hipfftHandle plan;
    CUFFT_SAFE_CALL(hipfftPlan1d(&plan, XN, HIPFFT_Z2Z, 1));

    // Create wave number
	double dkx = 2*M_PI/XN/DX;
	for(int i = XN/2; i >= 0; i--) 
		h_kx[XN/2 - i]=(XN/2 - i) * dkx;
	for(int i = XN/2+1; i < XN; i++) 
		h_kx[i]=(i - XN) * dkx; 

	// Initial conditions on host
	for(int i = 0; i < XN; i++)
		{
			h_x[i] = (i-XN/2)*DX;
			h_psi[i].x = sqrt(2)/cosh(h_x[i]);
			//h_psi[i].x = 2*exp(-(x[i]*x[i]/2.0/2.0));
			h_psi[i].y = 0;
			h_psi_0[i].x = h_psi[i].x;
			h_psi_0[i].y = h_psi[i].y;
			h_k2[i] = h_kx[i]*h_kx[i];
		}   
	
	// Allocate device arrays and copy from host
    hipfftDoubleComplex *d_psi; double *d_k2;
	CUDAR_SAFE_CALL(hipMalloc(&d_psi, sizeof(hipfftDoubleComplex)*XN));
	CUDAR_SAFE_CALL(hipMalloc(&d_k2, sizeof(double)*XN));
    CUDAR_SAFE_CALL(hipMemcpy(d_psi, h_psi, sizeof(hipfftDoubleComplex)*XN, hipMemcpyHostToDevice));
    CUDAR_SAFE_CALL(hipMemcpy(d_k2, h_k2, sizeof(double)*XN, hipMemcpyHostToDevice));
	
	// Initialize the grid
	dim3 threadsPerBlock(128,1,1);
	dim3 blocksPerGrid((XN + 127)/128,1,1);

	// Forward transform 
	CUFFT_SAFE_CALL(hipfftExecZ2Z(plan, d_psi, d_psi, HIPFFT_FORWARD));
	
	// Timing starts here
	hipEventRecord(begin_event, 0);
	
	// Start time evolution
	for (int i = 1; i <= TN; i++)
	{
		// Solve linear part
		lin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, d_k2, DT/2, XN);  
		#if CUDAR_ERROR_CHECKING
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		#endif // CUDAR_ERROR_CHECKING
		// Backward transform
    	CUFFT_SAFE_CALL(hipfftExecZ2Z(plan, d_psi, d_psi, HIPFFT_BACKWARD));
		// Normalize the transform
		normalize<<<blocksPerGrid, threadsPerBlock>>>(d_psi, XN);
		#if CUDAR_ERROR_CHECKING
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		#endif // CUDAR_ERROR_CHECKING
		// Solve nonlinear part
		nonlin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, DT, XN);
		#if CUDAR_ERROR_CHECKING
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		#endif // CUDAR_ERROR_CHECKING
		// Forward transform
    	CUFFT_SAFE_CALL(hipfftExecZ2Z(plan, d_psi, d_psi, HIPFFT_FORWARD));
		// Solve linear part
		lin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, d_k2, DT/2, XN);  
		#if CUDAR_ERROR_CHECKING
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		#endif // CUDAR_ERROR_CHECKING
	}
	float time_value;	
	hipEventRecord(end_event, 0);
	hipEventSynchronize(end_event);
	hipEventElapsedTime(&time_value, begin_event, end_event);

	// Print time to file
	FILE *fp = fopen(TIME_F, "a");
	fprintf(fp, "%f, ", time_value);
	fclose(fp);
	
	// Backward tranform to retreive data
	CUFFT_SAFE_CALL(hipfftExecZ2Z(plan, d_psi, d_psi, HIPFFT_BACKWARD));
	// Normalize the transform
	normalize<<<blocksPerGrid, threadsPerBlock>>>(d_psi, XN);
	#if CUDAR_ERROR_CHECKING
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		#endif // CUDAR_ERROR_CHECKING
	
	// Copy results to device
	CUDAR_SAFE_CALL(hipMemcpy(h_psi, d_psi, sizeof(hipfftDoubleComplex)*XN, 
															hipMemcpyDeviceToHost));
	// Plot results
	cm_plot_1d(h_psi_0, h_psi, LX, XN, PLOT_F);

	// Clean up
	CUFFT_SAFE_CALL(hipfftDestroy(plan));
	free(h_x);
	free(h_k2);
	free(h_kx);
    free(h_psi_0);
	free(h_psi);
	CUDAR_SAFE_CALL(hipFree(d_psi));
	CUDAR_SAFE_CALL(hipFree(d_k2));
	
	return 0;
}

__global__ void nonlin(hipfftDoubleComplex *psi, double dt, int xn)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
    
	// Avoid first and last point (boundary conditions) (needs fixing)
	//if (i >= xn - 1 || i == 0) return; 
	if (i >= xn) return; 
	
	double psi2 = hipCabs(psi[i])*hipCabs(psi[i]);
	psi[i] = hipCmul(psi[i], make_hipDoubleComplex(cos(psi2*dt), sin(psi2*dt)));
}

__global__ void lin(hipfftDoubleComplex *psi, double *k2, double dt, int xn)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	
	// Avoid first and last point (boundary conditions) (needs fixing)
	//if (i >= xn - 1 || i == 0) return; 
	if (i >= xn) return; 
    
	psi[i] = hipCmul(psi[i], make_hipDoubleComplex(cos(k2[i]*dt), -sin(k2[i]*dt)));
}

__global__ void normalize(hipfftDoubleComplex *psi, int size)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x; 

	// Stay within range since grid might be larger
	if (i >= size) return; 
	
	psi[i].x = psi[i].x/size; psi[i].y = psi[i].y/size;
}

