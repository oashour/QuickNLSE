#include "hip/hip_runtime.h"
/**********************************************************************************
* Numerical Solution for the Cubic Nonlinear Schrodinger Equation in (1+1)D	  	  *
* using explicit FDTD with second order splitting.                                *
* Coded by: Omar Ashour, Texas A&M University at Qatar, February 2015.    	      *
* ********************************************************************************/
#include "../lib/cu_helpers.h"

// Grid Parameters
#define XN	nodes				// number of spatial ndes
#define TN	100				// number of temporal nodes
#define L	10.0				// Spatial Period
#define TT	10.0                // Max time
#define DX	(2*L / XN)			// spatial step size
#define DT	(TT / TN)			// temporal step size

// Gaussian Pulse Parameters
#define A 1.0
#define R 2.0

// Output files
#define PLOT_F "gpu_fdtds_plot.m"
#define TIME_F argv[2]

__global__ void R_lin_kernel(float *Re, float *Im, float dt, int xn, float dx);
__global__ void I_lin_kernel(float *Re, float *Im, float dt, int xn, float dx);
__global__ void nonlin_kernel(float *Re, float *Im, float dt, int xn);

int main(int argc, char *argv[])
{
    // Timing info
	hipEvent_t begin_event, end_event;
	hipEventCreate(&begin_event);
	hipEventCreate(&end_event);
    
	// Print basic info about simulation
	const int nodes = atoi(argv[1]);
	printf("XN: %d. DX: %f, DT: %f, dt/dx^2: %f\n", XN, DX, DT, DT/(DX*DX));
	
	// Allocate host arrays
    float *h_x = (float*)malloc(sizeof(float) * XN);
	float *h_Re 	= (float*)malloc(sizeof(float) * XN);
    float *h_Im	= (float*)malloc(sizeof(float) * XN);   
	float *h_Re_0 	= (float*)malloc(sizeof(float) * XN);
    float *h_Im_0	= (float*)malloc(sizeof(float) * XN);   
	
	// Initial conditions on host
	for(int i = 0; i < XN ; i++)
	{
		h_x[i] = (i-XN/2)*DX;
		h_Re[i]	= sqrt(2.0)/(cosh(h_x[i]));	
		h_Im[i]	= 0;       		 				
		//h_Re[i]	= 2*exp(-(h_x[i]*h_x[i])/2.0/2.0);
		h_Im_0[i] = h_Im[i];
		h_Re_0[i] = h_Re[i];
	}
    
    // Allocate device arrays and copy from host.
	float *d_Re, *d_Im;
	CUDAR_SAFE_CALL(hipMalloc(&d_Re, sizeof(float) * XN));
	CUDAR_SAFE_CALL(hipMalloc(&d_Im, sizeof(float) * XN));
	CUDAR_SAFE_CALL(hipMemcpy(d_Re, h_Re, sizeof(float) * XN, hipMemcpyHostToDevice));
	CUDAR_SAFE_CALL(hipMemcpy(d_Im, h_Im, sizeof(float) * XN, hipMemcpyHostToDevice));

	// Initialize the grid
	dim3 threadsPerBlock(128,1,1);
	dim3 blocksPerGrid((XN + 127)/128,1,1);

	// Timing starts here
	hipEventRecord(begin_event, 0);
	
	// Start time evolution
	for (int i = 1; i <= TN; i++)
	{
		// Solve linear part
		R_lin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT*0.5, XN, DX);
		#if CUDAR_ERROR_CHECKING
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		#endif // CUDAR_ERROR_CHECKING
        I_lin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT*0.5, XN, DX);
		#if CUDAR_ERROR_CHECKING
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		#endif // CUDAR_ERROR_CHECKING
		// Solve nonlinear part
		nonlin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT, XN);
		#if CUDAR_ERROR_CHECKING
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		#endif // CUDAR_ERROR_CHECKING
		// Solve linear part
		R_lin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT*0.5, XN, DX);
		#if CUDAR_ERROR_CHECKING
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		#endif // CUDAR_ERROR_CHECKING
        I_lin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT*0.5, XN, DX);
		#if CUDAR_ERROR_CHECKING
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		#endif // CUDAR_ERROR_CHECKING
	}
	float time_value;	
	hipEventRecord(end_event, 0);
	hipEventSynchronize(end_event);
	hipEventElapsedTime(&time_value, begin_event, end_event);

	// Print time to file
	FILE *fp = fopen(TIME_F, "a");
	fprintf(fp, "%f, ", time_value);
	fclose(fp);
 
	// Copy results to device
	CUDAR_SAFE_CALL(hipMemcpy(h_Re, d_Re, sizeof(float)*XN, 
															hipMemcpyDeviceToHost));
	CUDAR_SAFE_CALL(hipMemcpy(h_Im, d_Im, sizeof(float)*XN, 
															hipMemcpyDeviceToHost));
	
	// PLot results
	m_plot_1df(h_Re_0, h_Im_0, h_Re, h_Im, L, XN, PLOT_F);
	
	// Clean up
	free(h_Re); 
	free(h_Im); 
	free(h_Re_0); 
	free(h_Im_0); 
	free(h_x); 
	CUDAR_SAFE_CALL(hipFree(d_Re)); 
	CUDAR_SAFE_CALL(hipFree(d_Im)); 

	return 0;
}

__global__ void R_lin_kernel(float *Re, float *Im, float dt, int xn, float dx)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	
	// Avoid first and last point (boundary conditions)
	if (i >= xn - 1 || i == 0) return; 

	Re[i] = Re[i] - dt/(dx*dx)*(Im[i+1] - 2*Im[i] + Im[i-1]);
}

__global__ void I_lin_kernel(float *Re, float *Im, float dt, int xn, float dx)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	
	// Avoid first and last point (boundary conditions)
	if (i >= xn - 1 || i == 0) return; 

	Im[i] = Im[i] + dt/(dx*dx)*(Re[i+1] - 2*Re[i] + Re[i-1]);
}

__global__ void nonlin_kernel(float *Re, float *Im, float dt, int xn)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	// Avoid first and last point (boundary conditions)
	if (i >= xn - 1 || i == 0) return; 
	
	float Rp = Re[i]; float Ip = Im[i];
	float A2 = Rp*Rp+Ip*Ip;
	
	Re[i] =	Rp*cos(A2*dt) - Ip*sin(A2*dt);
	Im[i] =	Rp*sin(A2*dt) + Ip*cos(A2*dt);
}

