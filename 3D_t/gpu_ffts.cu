#include "hip/hip_runtime.h"
/**********************************************************************************
* Numerical Solution for the Cubic-Quintic Nonlinear Schrodinger Equation         *
* using second order split step Fourier method.                                   *
* Coded by: Omar Ashour, Texas A&M University at Qatar, February 2015.    	      *
* ********************************************************************************/
#include <stddef.h>
#include "../lib/cu_helpers.h"
#include <hipfft/hipfft.h>

// Grid Parameters
#define XN	32						// Number of x-spatial nodes        
#define YN	32						// Number of y-spatial nodes          
#define ZN	32						// Number of z-spatial nodes         
#define TN	1000					// Number of temporal nodes          
#define LX	50.0f					// x-spatial domain [-LX,LX)         
#define LY	50.0f					// y-spatial domain [-LY,LY)         
#define LZ	50.0f					// z-spatial domain [-LZ,LZ)         
#define TT	10.0f            		// Max time                          
#define DX	(2*LX / XN)				// x-spatial step size               
#define DY	(2*LY / YN)				// y-spatial step size
#define DZ	(2*LZ / ZN)				// z-spatial step size
#define DT	(TT / TN)    			// temporal step size

// Gaussian Parameters                                     
#define  A_S 	(3.0f/sqrt(8.0f))
#define  R_S 	(sqrt(32.0f/9.0f))
#define  A 		0.6f
#define  R 		(1.0f/(A*sqrt(1.0f-A*A)))   
                                                                          
// Index flattening macro
// Flat[x + WIDTH * (y + DEPTH * z)] = Original[x, y, z]                  
#define ind(i,j,k) ((((i * ZN) * YN) + (j * YN)) + k)
//		   		 ____WIDTH____  
//		   		|_|_|_|_|_|_|_|H
//		   	 	|_|_|_|_|_|_|_|E
//		   	   Z|_|_|_|_|_|_|_|I
//		   	   N|_|_|_|_|_|_|_|G
//		   		|_|_|_|_|_|_|_|H
//		   	    |_|_|_|_|_|_|_|T
//		   	    \_\_\_\_\_\_\_\D
//               \_\_\_\_\_\_\_\E
//               Y\_\_\_\_\_\_\_\P
//                N\_\_\_\_\_\_\_\T
//					\_\_\_\_\_\_\_\H             
// 						  XN                          

// Timing parameters
#define IRVL	10				// Timing interval. Take a reading every N iterations.

// Output files
#define VTK_0  "gpu_ffts_0.vtk" 
#define VTK_1  "gpu_ffts_1.vtk"
#define TIME_F "gpu_ffts_time.m"

// Function prototypes
__global__ void nonlin(hipfftComplex *psi, float dt, int xn, int yn, int zn);
__global__ void lin(hipfftComplex *psi, float *k2, float dt, int xn, int yn, int zn);
__global__ void normalize(hipfftComplex *psi, int size, int xn, int yn, int zn);

int main(void)
{                                                                          
    // Timing info
	hipEvent_t begin_event, end_event;
	hipEventCreate(&begin_event);
	hipEventCreate(&end_event);
    
	// Print basic info about simulation
	printf("XN: %d. DX: %f, DT: %f, dt/dx^2: %f\n", XN, DX, DT, DT/(DX*DX));

	// Allocate host arrays
    float *h_x = (float*)malloc(sizeof(float) * XN);
	float *h_y = (float*)malloc(sizeof(float) * YN);
	float *h_z = (float*)malloc(sizeof(float) * ZN);
	float *h_k2 = (float*)malloc(sizeof(float) * XN * YN * ZN);
	float *h_kx = (float*)malloc(XN * sizeof(float));
	float *h_ky = (float*)malloc(YN * sizeof(float));
	float *h_kz = (float*)malloc(ZN * sizeof(float));
	float *h_max = (float*)calloc(TN+1, sizeof(float));
	hipfftComplex *h_psi = (hipfftComplex*)malloc(
							sizeof(hipfftComplex) * XN * YN * ZN);
	hipfftComplex *h_psi_0 = (hipfftComplex*)malloc(
							sizeof(hipfftComplex) * XN * YN * ZN);
	
	// Create transform plans
    hipfftHandle plan;
    CUFFT_SAFE_CALL(hipfftPlan3d(&plan, XN, YN, ZN, HIPFFT_C2C));

    // Create wavenumbers
	float dkx = 2*M_PI/XN/DX;
	for(int i = XN/2; i >= 0; i--) 
		h_kx[XN/2 - i]=(XN/2 - i) * dkx;
	for(int i = XN/2+1; i < XN; i++) 
		h_kx[i]=(i - XN) * dkx; 

	float dky = 2*M_PI/YN/DY;
	for(int i = YN/2; i >= 0; i--) 
		h_ky[YN/2 - i]=(YN/2 - i) * dky;
	for(int i = YN/2+1; i < YN; i++) 
		h_ky[i]=(i - YN) * dky; 

	float dkz = 2*M_PI/ZN/DZ;
	for(int i = ZN/2; i >= 0; i--) 
		h_kz[ZN/2 - i]=(ZN/2 - i) * dkz;
	for(int i = ZN/2+1; i < ZN; i++) 
		h_kz[i]=(i - ZN) * dkz; 
	
	// Initialize x, y and z
	for(int i = 0; i < XN ; i++)
		h_x[i] = (i-XN/2)*DX;
    
	for(int i = 0; i < YN ; i++)
		h_y[i] = (i-YN/2)*DY;

	for(int i = 0; i < ZN ; i++)
		h_z[i] = (i-ZN/2)*DZ;
	
	// Initial conditions on host
	for(int i = 0; i < XN; i++)
		for(int j = 0; j < YN; j++)
			for(int k = 0; k < ZN; k++)
			{
				h_psi[ind(i,j,k)].x = A_S*A*
							   		  exp(-(h_x[i]*h_x[i]+h_y[j]*h_y[j]+h_z[k]*h_z[k])
															/(2*R*R*R_S*R_S));
				h_psi[ind(i,j,k)].y = 0;
				h_psi_0[ind(i,j,k)].x = h_psi[ind(i,j,k)].x;
				h_psi_0[ind(i,j,k)].y = h_psi[ind(i,j,k)].y;
				h_k2[ind(i,j,k)] = h_kx[i]*h_kx[i] + h_ky[j]*h_ky[j] + h_kz[k]*h_kz[k];
			}   
	
	// Allocate and copy device memory
    hipfftComplex *d_psi; float *d_k2;
	CUDAR_SAFE_CALL(hipMalloc((void **)&d_psi, sizeof(hipfftComplex)*XN*YN*ZN));
	CUDAR_SAFE_CALL(hipMalloc((void **)&d_k2, sizeof(float)*XN*YN*ZN));
    CUDAR_SAFE_CALL(hipMemcpy(d_psi, h_psi, sizeof(hipfftComplex)*XN*YN*ZN,
															hipMemcpyHostToDevice));
    CUDAR_SAFE_CALL(hipMemcpy(d_k2, h_k2, sizeof(float)*XN*YN*ZN, 
															hipMemcpyHostToDevice));
	
	// Initialize the grid
	dim3 threadsPerBlock(8,8,8);
	dim3 blocksPerGrid((XN + 7)/8,(YN+7)/8,(ZN+7)/8);
	
	// Find max(|psi|) for initial pulse.
	//cmax_psi(psi, h_max, 0, XN*YN*ZN);
	
	// Print timing info to file
	float time_value;
	FILE *fp = fopen(TIME_F, "w");
	fprintf(fp, "steps = [0:%d:%d];\n", IRVL, TN);
	fprintf(fp, "time = [0, ");

	// Forward transform 
	CUFFT_SAFE_CALL(hipfftExecC2C(plan, d_psi, d_psi, HIPFFT_FORWARD));
	
	// Timing starts here
	hipEventRecord(begin_event, 0);
	for (int i = 1; i <= TN; i++)
	{
		// Solve linear part
		lin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, d_k2, DT/2, XN, YN, ZN);  
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// Backward transform
    	CUFFT_SAFE_CALL(hipfftExecC2C(plan, d_psi, d_psi, HIPFFT_BACKWARD));
		// Normalize the transform
		normalize<<<blocksPerGrid, threadsPerBlock>>>(d_psi, XN*YN*ZN, XN, YN, ZN);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// Solve nonlinear part 
		nonlin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, DT, XN, YN, ZN);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// Forward transform
    	CUFFT_SAFE_CALL(hipfftExecC2C(plan, d_psi, d_psi, HIPFFT_FORWARD));
		// Linear calculation
		lin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, d_k2, DT/2, XN, YN, ZN);  
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// Save max |psi| for printing
		//cmax_psi(psi, h_max, i, XN*YN*ZN);
		// Print time at specific intervals
		if(i % IRVL == 0)
		{
			hipEventRecord(end_event, 0);
			hipEventSynchronize(end_event);
			hipEventElapsedTime(&time_value, begin_event, end_event);
			fprintf(fp, "%f, ", time_value);
		}

	}
	// Wrap up timing file 
	fprintf(fp, "];\n");
	fprintf(fp, "plot(steps, time/1000, '-*r');\n");
	fclose(fp);
	
	// Backward tranform to retreive data
	CUFFT_SAFE_CALL(hipfftExecC2C(plan, d_psi, d_psi, HIPFFT_BACKWARD));
	// Normalize the transform
	normalize<<<blocksPerGrid, threadsPerBlock>>>(d_psi, XN*YN*ZN, XN, YN, ZN);
	CUDAR_SAFE_CALL(hipPeekAtLastError());
	
	// Copy results to device
	CUDAR_SAFE_CALL(hipMemcpy(h_psi, d_psi, sizeof(hipfftComplex)*XN*YN*ZN, 
															hipMemcpyDeviceToHost));
	// Plot results
	vtk_3dcf(h_x, h_y, h_z, h_psi, XN, YN, ZN, VTK_1);
	vtk_3dcf(h_x, h_y, h_z, h_psi_0, XN, YN, ZN, VTK_0);

	// Clean up 
	CUFFT_SAFE_CALL(hipfftDestroy(plan));
	free(h_x);
	free(h_y);
	free(h_z);
	free(h_k2);
	free(h_kx);
	free(h_ky);
	free(h_kz);
	free(h_psi);
	free(h_psi_0);
	free(h_max);
	CUDAR_SAFE_CALL(hipFree(d_psi));
	CUDAR_SAFE_CALL(hipFree(d_k2));
	
	return 0;
}

__global__ void nonlin(hipfftComplex *psi, float dt, int xn, int yn, int zn)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	int j = threadIdx.y + blockIdx.y * blockDim.y; 
	int k = threadIdx.z + blockIdx.z * blockDim.z; 

	// Avoid first and last point (boundary conditions) (needs fixing)
	// if (i >= xn - 1 || j >= yn-1 || || k >= zn-1 || i == 0 || j == 0 || k == 0) return; 
    if (i >= xn || j >= yn || k >= zn) return;

	float psi2 = hipCabsf(psi[ind(i,j,k)])*hipCabsf(psi[ind(i,j,k)]);
    float non = psi2 - psi2*psi2;
	psi[ind(i,j,k)] = hipCmulf(psi[ind(i,j,k)], 
							make_hipComplex(cos(non*dt), sin(non*dt)));
}

__global__ void lin(hipfftComplex *psi, float *k2, float dt, int xn, int yn, int zn)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	int j = threadIdx.y + blockIdx.y * blockDim.y; 
	int k = threadIdx.z + blockIdx.z * blockDim.z; 
    
	// Avoid first and last point (boundary conditions) (needs fixing)
	// if (i >= xn - 1 || j >= yn-1 || || k >= zn-1 || i == 0 || j == 0 || k == 0) return; 
    if (i >= xn || j >= yn || k >= zn) return;
	
	psi[ind(i,j,k)] = hipCmulf(psi[ind(i,j,k)], 
				make_hipComplex(cos(k2[ind(i,j,k)]*dt), -sin(k2[ind(i,j,k)]*dt)));
}

__global__ void normalize(hipfftComplex *psi, int size, int xn, int yn, int zn)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	int j = threadIdx.y + blockIdx.y * blockDim.y; 
	int k = threadIdx.z + blockIdx.z * blockDim.z; 

	// Stay within range since the grid might be larger
    if (i >= xn || j >= yn || k >= zn) return;
	
	psi[ind(i,j,k)].x = psi[ind(i,j,k)].x/size; 
	psi[ind(i,j,k)].y = psi[ind(i,j,k)].y/size;
}

