#include "hip/hip_runtime.h"
/**********************************************************************************
* Numerical Solution for the Cubic-Quintic Nonlinear Schrodinger Equation         *
* using second order split step Fourier method.                                   *
* Coded by: Omar Ashour, Texas A&M University at Qatar, February 2015.    	      *
* ********************************************************************************/
#include <stddef.h>
#include "../lib/cu_helpers.h"
#include <hipfft/hipfft.h>

// Grid Parameters
#define XN	256						// Number of x-spatial nodes        
#define YN	256						// Number of y-spatial nodes          
#define ZN	256						// Number of z-spatial nodes         
#define TN	1000					// Number of temporal nodes          
#define LX	50.0					// x-spatial domain [-LX,LX)         
#define LY	50.0					// y-spatial domain [-LY,LY)         
#define LZ	50.0					// z-spatial domain [-LZ,LZ)         
#define TT	100.0            		// Max time                          
#define DX	(2*LX / XN)				// x-spatial step size               
#define DY	(2*LY / YN)				// y-spatial step size
#define DZ	(2*LZ / ZN)				// z-spatial step size
#define DT	(TT / TN)    			// temporal step size

// Gaussian Parameters                                     
#define  A_S 	(3.0/sqrt(8.0))
#define  R_S 	(sqrt(32.0/9.0))
#define  A 		0.6
#define  R 		(1.0/(A*sqrt(1.0-A*A)))   
                                                                          
// Index linearization                                                    
// Flat[x + WIDTH * (y + DEPTH * z)] = Original[x, y, z]                  
#define ind(i,j,k) ((i) + XN * ((j) + YN * (k)))		                     
//		   		 ____WIDTH____  
//		   		|_|_|_|_|_|_|_|H
//		   	 	|_|_|_|_|_|_|_|E
//		   	   Z|_|_|_|_|_|_|_|I
//		   	   N|_|_|_|_|_|_|_|G
//		   		|_|_|_|_|_|_|_|H
//		   	    |_|_|_|_|_|_|_|T
//		   	    \_\_\_\_\_\_\_\D
//               \_\_\_\_\_\_\_\E
//               Y\_\_\_\_\_\_\_\P
//                N\_\_\_\_\_\_\_\T
//					\_\_\_\_\_\_\_\H             
// 						  XN                          

// Function prototypes
__global__ void nonlin(hipfftDoubleComplex *psi, double dt);
__global__ void lin(hipfftDoubleComplex *psi, double *k2, double dt);
__global__ void normalize(hipfftDoubleComplex *psi, int size);

int main(void)
{                                                                          
	// Allocate and initialize the arrays
	double *h_k2 = (double*)malloc(sizeof(double) * XN * YN);
	hipfftDoubleComplex *h_psi = (hipfftDoubleComplex*)malloc(
							sizeof(hipfftDoubleComplex) * XN * YN * ZN);
	hipfftDoubleComplex *h_psi_0 = (hipfftDoubleComplex*)malloc(
							sizeof(hipfftDoubleComplex) * XN * YN * ZN);
	
	// Create transform plans
    hipfftHandle plan;
    CUFFT_SAFE_CALL(hipfftPlan3d(&plan, XN, YN, ZN, HIPFFT_Z2Z));

    // X and Y wave numbers
	double dkx = 2*M_PI/XN/DX;
	double *h_kx = (double*)malloc(XN * sizeof(double));
	for(int i = XN/2; i >= 0; i--) 
		h_kx[XN/2 - i]=(XN/2 - i) * dkx;
	for(int i = XN/2+1; i < XN; i++) 
		h_kx[i]=(i - XN) * dkx; 

	double dky = 2*M_PI/YN/DY;
	double *h_ky = (double*)malloc(ZN * sizeof(double));
	for(int i = YN/2; i >= 0; i--) 
		h_ky[YN/2 - i]=(YN/2 - i) * dky;
	for(int i = YN/2+1; i < YN; i++) 
		h_ky[i]=(i - YN) * dky; 

	double dkz = 2*M_PI/ZN/DZ;
	double *h_kz = (double*)malloc(ZN * sizeof(double));
	for(int i = ZN/2; i >= 0; i--) 
		h_kz[ZN/2 - i]=(ZN/2 - i) * dkz;
	for(int i = ZN/2+1; i < ZN; i++) 
		h_kz[i]=(i - ZN) * dkz; 
	
	// initialize x and y.
    double *h_x = (double*)malloc(sizeof(double) * XN);
	double *h_y = (double*)malloc(sizeof(double) * YN);
	double *h_z = (double*)malloc(sizeof(double) * ZN);
	
	for(int i = 0; i < XN ; i++)
		h_x[i] = (i-XN/2)*DX;
    
	for(int i = 0; i < YN ; i++)
		h_y[i] = (i-YN/2)*DY;

	for(int i = 0; i < ZN ; i++)
		h_z[i] = (i-ZN/2)*DZ;
	
	// Initial Conditions and square of wave number
	double *k2 = (double*)malloc(sizeof(double) * XN * YN * ZN);
	
	for (int k = 0; k < ZN; k++)
    	for(int j = 0; j < YN; j++)
			for(int i = 0; i < XN; i++)
			{
				h_psi[ind(i,j,k)].x = A_S*A*
							   		  exp(-(h_x[i]*h_x[i]+h_y[j]*h_y[j]+h_z[k]*h_z[k])
															/(2*R*R*R_S*R_S));
				h_psi[ind(i,j,k)].y = 0;
				h_psi_0[ind(i,j,k)].x = h_psi[ind(i,j,k)].x;
				h_psi_0[ind(i,j,k)].y = h_psi[ind(i,j,k)].y;
				h_k2[ind(i,j,k)] = h_kx[i]*h_kx[i] + h_ky[j]*h_ky[j] + h_kz[k]*h_kz[k];
			}   
	
	// Allocate and copy device memory
    hipfftDoubleComplex *d_psi; double *d_k2, *d_max;
	CUDAR_SAFE_CALL(hipMalloc((void **)&d_psi, sizeof(hipfftDoubleComplex)*XN*YN));
	CUDAR_SAFE_CALL(hipMalloc((void **)&d_k2, sizeof(double)*XN*YN));
    CUDAR_SAFE_CALL(hipMemcpy(d_psi, h_psi, sizeof(hipfftDoubleComplex)*XN*YN*ZN,
															hipMemcpyHostToDevice));
    CUDAR_SAFE_CALL(hipMemcpy(d_k2, h_k2, sizeof(double)*XN*YN*ZN, 
															hipMemcpyHostToDevice));
	
	// initialize the grid
	dim3 threadsPerBlock(8,8,8);
	dim3 blocksPerGrid((XN + 7)/8,(YN+7)/8,(ZN+7)/8);
	
	// Find max(|psi|) for initial pulse.
	double *h_max = (double*)calloc(TN, sizeof(double));
	//double *h_max = (double*)malloc(sizeof(double) * TN);
	//cmax_psi(psi, max, 0, XN*YN);
	for (int i = 1; i < TN; i++)
	{
		// forward transform
    	CUFFT_SAFE_CALL(hipfftExecZ2Z(plan, d_psi, d_psi, HIPFFT_FORWARD));
		// linear calculation
		lin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, d_k2, DT/2);  
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// backward transform
    	CUFFT_SAFE_CALL(hipfftExecZ2Z(plan, d_psi, d_psi, HIPFFT_BACKWARD));
		// normalize the transform
		normalize<<<blocksPerGrid, threadsPerBlock>>>(d_psi, XN*YN*ZN);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// nonlinear calculation
		nonlin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, DT);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// forward transform
    	CUFFT_SAFE_CALL(hipfftExecZ2Z(plan, d_psi, d_psi, HIPFFT_FORWARD));
		// linear calculation
		lin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, d_k2, DT/2);  
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// backward transform
    	CUFFT_SAFE_CALL(hipfftExecZ2Z(plan, d_psi, d_psi, HIPFFT_BACKWARD));
		// normalize the transform
		normalize<<<blocksPerGrid, threadsPerBlock>>>(d_psi, XN*YN*ZN);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		//cmax_psi(d_psi, h_max, i, XN*YN);
	}
	CUDAR_SAFE_CALL(hipMemcpy(h_psi, d_psi, sizeof(hipfftDoubleComplex)*XN*YN*ZN, 
															hipMemcpyDeviceToHost));
	// plot results
	//cm_plot_2d(h_psi_0, h_psi, h_max, LX, LY, XN, YN, TN, "gpufft.m");

	// garbage collection
	CUFFT_SAFE_CALL(hipfftDestroy(plan));
	free(h_x);
	free(h_y);
	free(h_z);
	free(h_k2);
	free(h_kx);
	free(h_ky);
	free(h_kz);
	free(h_psi);
	free(h_psi_0);
	free(h_max);
	CUDAR_SAFE_CALL(hipFree(d_psi));
	CUDAR_SAFE_CALL(hipFree(d_k2));
	CUDAR_SAFE_CALL(hipFree(d_max));
	
	return 0;
}

__global__ void nonlin(hipfftDoubleComplex *psi, double dt)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	int j = threadIdx.y + blockIdx.y * blockDim.y; 
	int k = threadIdx.z + blockIdx.z * blockDim.z; 

    if (i >= XN || j >= YN || k >= ZN) return;

	double psi2 = hipCabs(psi[ind(i,j,k)])*hipCabs(psi[ind(i,j,k)]);
    double non = psi2 - psi2*psi2;
	hipfftDoubleComplex expo = make_hipDoubleComplex(cos(non*dt), sin(non*dt));
	psi[ind(i,j,k)] = hipCmul(psi[ind(i,j,k)], expo);
}

__global__ void lin(hipfftDoubleComplex *psi, double *k2, double dt)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	int j = threadIdx.y + blockIdx.y * blockDim.y; 
	int k = threadIdx.z + blockIdx.z * blockDim.z; 
    
    if (i >= XN || j >= YN || k >= ZN) return;
	
	hipfftDoubleComplex expo = make_hipDoubleComplex(
								cos(k2[ind(i,j,k)]*dt), -sin(k2[ind(i,j,k)]*dt));
	psi[ind(i,j,k)] = hipCmul(psi[ind(i,j,k)], expo);
}

__global__ void normalize(hipfftDoubleComplex *psi, int size)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	int j = threadIdx.y + blockIdx.y * blockDim.y; 
	int k = threadIdx.z + blockIdx.z * blockDim.z; 

    if (i >= XN || j >= YN || k >= ZN) return;
	
	psi[ind(i,j,k)].x = psi[ind(i,j,k)].x/size; 
	psi[ind(i,j,k)].y = psi[ind(i,j,k)].y/size;
}

