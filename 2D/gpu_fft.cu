#include "hip/hip_runtime.h"
/**********************************************************************************
* Numerical Solution for the Cubic-Quintic Nonlinear Schrodinger Equation         *
* using second order split step Fourier method.                                   *
* Coded by: Omar Ashour, Texas A&M University at Qatar, February 2015.    	      *
* ********************************************************************************/
#include <stddef.h>
#include "../lib/cu_helpers.h"
#include <hipfft/hipfft.h>

// Grid Parameters
#define XN	256				   		// Number of x-spatial nodes
#define YN	256						// Number of y-spatial nodes
#define TN	1000					// Number of temporal nodes
#define LX	50.0					// x-spatial domain [-LX,LX)
#define LY	50.0					// y-spatial domain [-LY,LY)
#define TT	10.0            		// Max time
#define DX	(2*LX / XN)				// x-spatial step size
#define DY	(2*LY / YN)				// y-spatial step size
#define DT	(TT / TN)    			// temporal step size

// Gaussian Parameters                                     
#define  A_S 	(3.0/sqrt(8.0))
#define  R_S 	(sqrt(32.0/9.0))
#define  A 		0.6
#define  R 		(1.0/(A*sqrt(1.0-A*A)))   

// Index linearization
#define ind(i,j)  (i*XN+j)			// [i  ,j  ] 

// Function prototypes
__global__ void nonlin(hipfftDoubleComplex *psi, double dt);
__global__ void lin(hipfftDoubleComplex *psi, double *k2, double dt);
__global__ void normalize(hipfftDoubleComplex *psi, int size);

int main(void)
{                                                                          
	// Allocate and initialize the arrays
    double *h_x = (double*)malloc(sizeof(double) * XN);
	double *h_y = (double*)malloc(sizeof(double) * YN);
	double *h_kx = (double*)malloc(sizeof(double)*XN);
	double *h_ky = (double*)malloc(sizeof(double)*YN);
	double *h_max = (double*)calloc(TN, sizeof(double));
	//double *h_max = (double*)malloc(sizeof(double) * TN);
	double *h_k2 = (double*)malloc(sizeof(double) * XN * YN);
	hipfftDoubleComplex *h_psi = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex) * XN * YN);
	hipfftDoubleComplex *h_psi_0 = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex) * XN * YN);
	
	// Create transform plans
    hipfftHandle plan;
    CUFFT_SAFE_CALL(hipfftPlan2d(&plan, XN, YN, HIPFFT_Z2Z));

    // X and Y wave numbers
	double dkx = 2*M_PI/XN/DX;
	for(int i = XN/2; i >= 0; i--) 
		h_kx[XN/2 - i]=(XN/2 - i) * dkx;
	for(int i = XN/2+1; i < XN; i++) 
		h_kx[i]=(i - XN) * dkx; 

	double dky = 2*M_PI/YN/DY;
	for(int i = YN/2; i >= 0; i--) 
		h_ky[YN/2 - i]=(YN/2 - i) * dky;
	for(int i = YN/2+1; i < YN; i++) 
		h_ky[i]=(i - YN) * dky; 

	// initialize x and y.
	for(int i = 0; i < XN ; i++)
		h_x[i] = (i-XN/2)*DX;
    
	for(int i = 0; i < YN ; i++)
		h_y[i] = (i-YN/2)*DY;

	// Initial Conditions and square of wave number
    for(int j = 0; j < YN; j++)
		for(int i = 0; i < XN; i++)
			{
				h_psi[ind(i,j)].x = A_S*A*exp(-(h_x[i]*h_x[i]+h_y[j]*h_y[j])
															/(2*R*R*R_S*R_S));
				h_psi[ind(i,j)].y = 0;
				h_psi_0[ind(i,j)].x = h_psi[ind(i,j)].x;
				h_psi_0[ind(i,j)].y = h_psi[ind(i,j)].y;
				h_k2[ind(i,j)] = h_kx[i]*h_kx[i] + h_ky[j]*h_ky[j];
			}   
	
	// Allocate and copy device memory
    hipfftDoubleComplex *d_psi; double *d_k2, *d_max;
	CUDAR_SAFE_CALL(hipMalloc((void **)&d_psi, sizeof(hipfftDoubleComplex)*XN*YN));
	CUDAR_SAFE_CALL(hipMalloc((void **)&d_k2, sizeof(double)*XN*YN));
	CUDAR_SAFE_CALL(hipMalloc((void **)&d_max, sizeof(double)*TN));
    CUDAR_SAFE_CALL(hipMemcpy(d_psi, h_psi, sizeof(hipfftDoubleComplex)*XN*YN, hipMemcpyHostToDevice));
    CUDAR_SAFE_CALL(hipMemcpy(d_k2, h_k2, sizeof(double)*XN*YN, hipMemcpyHostToDevice));
	
	// initialize the grid
	dim3 threadsPerBlock(16,16,1);
	dim3 blocksPerGrid((XN + 15)/16,(YN+15)/16,1);
	
	// Find max(|psi|) for initial pulse.
	//cmax_psi(psi, max, 0, XN*YN);
	for (int i = 1; i < TN; i++)
	{
		// forward transform
    	CUFFT_SAFE_CALL(hipfftExecZ2Z(plan, d_psi, d_psi, HIPFFT_FORWARD));
		// linear calculation
		lin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, d_k2, DT/2);  
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// backward transform
    	CUFFT_SAFE_CALL(hipfftExecZ2Z(plan, d_psi, d_psi, HIPFFT_BACKWARD));
		// normalize the transform
		normalize<<<blocksPerGrid, threadsPerBlock>>>(d_psi, XN*YN);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// nonlinear calculation
		nonlin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, DT);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// forward transform
    	CUFFT_SAFE_CALL(hipfftExecZ2Z(plan, d_psi, d_psi, HIPFFT_FORWARD));
		// linear calculation
		lin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, d_k2, DT/2);  
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// backward transform
    	CUFFT_SAFE_CALL(hipfftExecZ2Z(plan, d_psi, d_psi, HIPFFT_BACKWARD));
		// normalize the transform
		normalize<<<blocksPerGrid, threadsPerBlock>>>(d_psi, XN*YN);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		//cmax_psi(d_psi, d_max, 0, XN*YN);
	}
	CUDAR_SAFE_CALL(hipMemcpy(h_psi, d_psi, sizeof(hipfftDoubleComplex)*XN*YN, hipMemcpyDeviceToHost));
	//CUDAR_SAFE_CALL(hipMemcpy(h_max, d_max, sizeof(double)*TN, hipMemcpyDeviceToHost));
	// plot results
	cm_plot_2d(h_psi_0, h_psi, h_max, LX, LY, XN, YN, TN, "gpufft.m");

	// garbage collection
	CUFFT_SAFE_CALL(hipfftDestroy(plan));
	free(h_x);
	free(h_y);
	free(h_k2);
	free(h_kx);
	free(h_ky);
	free(h_psi);
	free(h_psi_0);
	free(h_max);
	CUDAR_SAFE_CALL(hipFree(d_psi));
	CUDAR_SAFE_CALL(hipFree(d_k2));
	CUDAR_SAFE_CALL(hipFree(d_max));
	
	return 0;
}

__global__ void nonlin(hipfftDoubleComplex *psi, double dt)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	int j = threadIdx.y + blockIdx.y * blockDim.y; 
	
    if (i >= XN || j >= YN) return;

	double psi2 = hipCabs(psi[ind(i,j)])*hipCabs(psi[ind(i,j)]);
    double non = psi2 - psi2*psi2;
	hipfftDoubleComplex expo = make_hipDoubleComplex(cos(non*dt), sin(non*dt));
	psi[ind(i,j)] = hipCmul(psi[ind(i,j)], expo);
}

__global__ void lin(hipfftDoubleComplex *psi, double *k2, double dt)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	int j = threadIdx.y + blockIdx.y * blockDim.y; 
    
    if (i >= XN || j >= YN) return;
	
	hipfftDoubleComplex expo = make_hipDoubleComplex(
								cos(k2[ind(i,j)]*dt), -sin(k2[ind(i,j)]*dt));
	psi[ind(i,j)] = hipCmul(psi[ind(i,j)], expo);
}

__global__ void normalize(hipfftDoubleComplex *psi, int size)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	int j = threadIdx.y + blockIdx.y * blockDim.y; 

    if (i >= XN || j >= YN) return;
	
	psi[ind(i,j)].x = psi[ind(i,j)].x/size; psi[ind(i,j)].y = psi[ind(i,j)].y/size;
}

