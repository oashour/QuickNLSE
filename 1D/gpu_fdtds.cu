#include "hip/hip_runtime.h"
// nlse
#include "../lib/cu_helpers.h"

// given stuff
#define XN	1000
#define TN	100000
#define L	10.0
#define TT	10.0

// calculated from given
#define DX	(2*L / XN)
#define DT	(TT / TN)

// Gaussian Pulse Parameters
#define A 1.0
#define R 2.0

__global__ void R_lin_kernel(float *Re, float *Im, float dt);
__global__ void I_lin_kernel(float *Re, float *Im, float dt);
__global__ void nonlin_kernel(float *Re, float *Im, float dt);

int main(void)
{
    printf("DX: %f, DT: %f, dt/dx^2: %f\n", DX, DT, DT/(DX*DX));

    hipEvent_t beginEvent;
	hipEvent_t endEvent;
 
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	
	// create the arrays x and t
    float *h_x = (float*)malloc(sizeof(float) * XN);
    // create used arrays
	float *h_Re 	= (float*)malloc(sizeof(float) * XN);
    float *h_Im	= (float*)malloc(sizeof(float) * XN);   
	float *h_Re_0 	= (float*)malloc(sizeof(float) * XN);
    float *h_Im_0	= (float*)malloc(sizeof(float) * XN);   
	// initial conditions.
	for(int i = 0; i < XN ; i++)
	{
		h_x[i] = (i-XN/2)*DX;
		h_Re[i]	= sqrt(2.0)/(cosh(h_x[i]));	// initial
		h_Im[i]	= 0;       		 				// initial
		//h_Re[i]	= 2*exp(-(h_x[i]*h_x[i])/2.0/2.0);	// initial
		h_Im_0[i] = h_Im[i];
		h_Re_0[i] = h_Re[i];
	}
    
    // allocate arrays on device and copy them
	float *d_Re, *d_Im;
	CUDAR_SAFE_CALL(hipMalloc(&d_Re, sizeof(float) * XN));
	CUDAR_SAFE_CALL(hipMalloc(&d_Im, sizeof(float) * XN));
	CUDAR_SAFE_CALL(hipMemcpy(d_Re, h_Re, sizeof(float) * XN, hipMemcpyHostToDevice));
	CUDAR_SAFE_CALL(hipMemcpy(d_Im, h_Im, sizeof(float) * XN, hipMemcpyHostToDevice));

	// initialize the grid
	dim3 threadsPerBlock(128,1,1);
	dim3 blocksPerGrid((XN + 127)/128,1,1);

	// solve 
	hipEventRecord(beginEvent, 0);
	for (int i = 1; i < TN; i++)
	{
		// linear
		R_lin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT*0.5);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
        I_lin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT*0.5);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// nonlinear
		nonlin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// linear
		R_lin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT*0.5);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
        I_lin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT*0.5);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
	}
	hipEventRecord(endEvent, 0);
    hipEventSynchronize(endEvent);
	float timeValue;
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	printf("%f\n", timeValue/1000.0);

	CUDAR_SAFE_CALL(hipMemcpy(h_Re, d_Re, sizeof(float)*XN, 
															hipMemcpyDeviceToHost));
	CUDAR_SAFE_CALL(hipMemcpy(h_Im, d_Im, sizeof(float)*XN, 
															hipMemcpyDeviceToHost));
	m_plot_1df(h_Re_0, h_Im_0, h_Re, h_Im, L, XN, "gpu_fdtd.m");
	// wrap up
	free(h_Re); 
	free(h_Im); 
	free(h_Re_0); 
	free(h_Im_0); 
	free(h_x); 
	CUDAR_SAFE_CALL(hipFree(d_Re)); 
	CUDAR_SAFE_CALL(hipFree(d_Im)); 

	return 0;
}

__global__ void R_lin_kernel(float *Re, float *Im, float dt)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	
	if (i >= XN - 1 || i == 0) return; // avoid first and last elements

	Re[i] = Re[i] - dt/(DX*DX)*(Im[i+1] - 2*Im[i] + Im[i-1]);
}

__global__ void I_lin_kernel(float *Re, float *Im, float dt)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	
	if (i >= XN - 1 || i == 0) return; // avoid first and last elements

	Im[i] = Im[i] + dt/(DX*DX)*(Re[i+1] - 2*Re[i] + Re[i-1]);
}

__global__ void nonlin_kernel(float *Re, float *Im, float dt)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	float Rp = Re[i]; float Ip = Im[i];
	float A2 = Rp*Rp+Ip*Ip;
	
	if (i > XN - 1) return; 
	
	Re[i] =	Rp*cos(A2*dt) - Ip*sin(A2*dt);
	Im[i] =	Rp*sin(A2*dt) + Ip*cos(A2*dt);
}

