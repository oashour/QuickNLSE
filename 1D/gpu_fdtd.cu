#include "hip/hip_runtime.h"
/**********************************************************************************
* Numerical Solution for the Cubic Nonlinear Schrodinger Equation in (1+1)D	  	  *
* using explicit FDTD with second order splitting.                                *
* Coded by: Omar Ashour, Texas A&M University at Qatar, February 2015.    	      *
* ********************************************************************************/
#include "../lib/cu_helpers.h"

// Grid Parameters
#define XN	1024				// number of spatial ndes
#define TN	100000				// number of temporal nodes
#define L	10.0				// Spatial Period
#define TT	10.0                // Max time
#define DX	(2*L / XN)			// spatial step size
#define DT	(TT / TN)			// temporal step size

// Gaussian Pulse Parameters
#define A 1.0
#define R 2.0

// Function Prototypes
__global__ void Re_lin_kernel(double *Re, double *Im, double dt, int xn, double dx);
__global__ void Im_lin_kernel(double *Re, double *Im, double dt, int xn, double dx);
__global__ void nonlin_kernel(double *Re, double *Im, double dt, int xn);

int main(void)
{
    // Timing info
	hipEvent_t begin_event, end_event;
	hipEventCreate(&begin_event);
	hipEventCreate(&end_event);
    
	// Timing starts here
	hipEventRecord(beginEvent, 0);
	
	// Print basic info about simulation
	printf("XN: %d. DX: %f, DT: %f, dt/dx^2: %f\n", XN, DX, DT, DT/(DX*DX));

	// Allocate host arrays
    double *h_x = (double*)malloc(sizeof(double) * XN);
	double *h_Re 	= (double*)malloc(sizeof(double) * XN);
    double *h_Im	= (double*)malloc(sizeof(double) * XN);   
	double *h_Re_0 	= (double*)malloc(sizeof(double) * XN);
    double *h_Im_0	= (double*)malloc(sizeof(double) * XN);   
	
	// Initial conditions on host
	for(int i = 0; i < XN ; i++)
	{
		h_x[i] = (i-XN/2)*DX;
		h_Re[i]	= sqrt(2.0)/(cosh(h_x[i]));	
		h_Im[i]	= 0;       		 			
		//h_Re[i]	= 2*exp(-(h_x[i]*h_x[i])/2.0/2.0);	
		h_Im_0[i] = h_Im[i];
		h_Re_0[i] = h_Re[i];
	}
    
    // Allocate device arrays on and copy from host
	double *d_Re, *d_Im;
	CUDAR_SAFE_CALL(hipMalloc(&d_Re, sizeof(double) * XN));
	CUDAR_SAFE_CALL(hipMalloc(&d_Im, sizeof(double) * XN));
	CUDAR_SAFE_CALL(hipMemcpy(d_Re, h_Re, sizeof(double) * XN, hipMemcpyHostToDevice));
	CUDAR_SAFE_CALL(hipMemcpy(d_Im, h_Im, sizeof(double) * XN, hipMemcpyHostToDevice));

	// Initialize the grid
	dim3 threadsPerBlock(128,1,1);
	dim3 blocksPerGrid((XN + 127)/128,1,1);

	// Start time evolution
	for (int i = 1; i < TN; i++)
	{
		// Solve linear part
		Re_lin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT*0.5);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
        Im_lin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT*0.5);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// Solve nonlinear part
		nonlin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
		// Solve linear part
		Re_lin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT*0.5);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
        Im_lin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Re, d_Im, DT*0.5);
		CUDAR_SAFE_CALL(hipPeekAtLastError());
	}
	
	// End timing and print to file 
	hipEventRecord(end_event, 0);
    hipEventSynchronize(end_event);
 
	float time_value;
	hipEventElapsedTime(&time_value, begin_event, end_event);
	
	FILE *fp = fopen(argv[2], "a");
	fprintf(fp, "%f, ", t2-t1);
	fclose(fp);

	// Copy results to device
	CUDAR_SAFE_CALL(hipMemcpy(h_Re, d_Re, sizeof(double)*XN, 
															hipMemcpyDeviceToHost));
	CUDAR_SAFE_CALL(hipMemcpy(h_Im, d_Im, sizeof(double)*XN, 
															hipMemcpyDeviceToHost));
	
	// Plot results
	m_plot_1d(h_Re_0, h_Im_0, h_Re, h_Im, L, XN, "gpu_fdtd.m");
	
	// Clean up 
	free(h_Re); 
	free(h_Im); 
	free(h_Re_0); 
	free(h_Im_0); 
	free(h_x); 
	CUDA_R_SAFECALL(hipFree(d_Re)); 
	CUDA_R_SAFECALL(hipFree(d_Im)); 

	return 0;
}

__global__ void Re_lin_kernel(double *Re, double *Im, double dt, int xn, double dx)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	
	// Avoid first and last point (boundary conditions)
	if (i >= xn - 1 || i == 0) return; 

	Re[i] = Re[i] - dt/(dx*dx)*(Im[i+1] - 2*Im[i] + Im[i-1]);
}

__global__ void Im_lin_kernel(double *Re, double *Im, double dt, int xn, double dx)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	
	// Avoid first and last point (boundary conditions)
	if (i >= xn - 1 || i == 0) return; 

	Im[i] = Im[i] + dt/(dx*dx)*(Re[i+1] - 2*Re[i] + Re[i-1]);
}

__global__ void nonlin_kernel(double *Re, double *Im, double dt, int xn)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	// Avoid first and last point (boundary conditions)
	if (i >= xn - 1 || i == 0) return; 

	double Rp = Re[i]; double Ip = Im[i];
	double A2 = Rp*Rp+Ip*Ip;
	
	Re[i] =	Rp*cos(A2*dt) - Ip*sin(A2*dt);
	Im[i] =	Rp*sin(A2*dt) + Ip*cos(A2*dt);
}

