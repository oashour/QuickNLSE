#include "hip/hip_runtime.h"
/**********************************************************************************
* Numerical Solution for the Cubic-Quintic Nonlinear Schrodinger Equation         *
* using second order split step Fourier method.                                   *
* Coded by: Omar Ashour, Texas A&M University at Qatar, February 2015.    	      *
**********************************************************************************/
#include <sys/time.h>
#include <stddef.h>
#include "../lib/cu_helpers.h"
#include <hipfft/hipfft.h>

// Grid Parameters
#define XN	2048					// Number of x-spatial nodes
#define TN	10000					// Number of temporal nodes
#define LX	10.0					// x-spatial domain [-LX,LX)
#define TT	10.0            		// Max time
#define DX	(2*LX / XN)				// x-spatial step size
#define DT	(TT / TN)    			// temporal step size

// Error checking Macros
// Macro to catch cufft errors
#define CUFFT_SAFE_CALL( call) do {                                            \
    hipfftResult err = call;                                                    \
    if (err != HIPFFT_SUCCESS) {                                                \
        fprintf(stderr, "CUFFT error in file '%s' in line %i: %s, %d.\n",	   \
                __FILE__, __LINE__, _cudaGetErrorEnum(err), err);          	   \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

static const char *_cudaGetErrorEnum(hipfftResult error)
{
    switch (error)
    {
        case HIPFFT_SUCCESS:
            return "HIPFFT_SUCCESS";

        case HIPFFT_INVALID_PLAN:
            return "HIPFFT_INVALID_PLAN";

        case HIPFFT_ALLOC_FAILED:
            return "HIPFFT_ALLOC_FAILED";

        case HIPFFT_INVALID_TYPE:
            return "HIPFFT_INVALID_TYPE";

        case HIPFFT_INVALID_VALUE:
            return "HIPFFT_INVALID_VALUE";

        case HIPFFT_INTERNAL_ERROR:
            return "HIPFFT_INTERNAL_ERROR";

        case HIPFFT_EXEC_FAILED:
            return "HIPFFT_EXEC_FAILED";

        case HIPFFT_SETUP_FAILED:
            return "HIPFFT_SETUP_FAILED";

        case HIPFFT_INVALID_SIZE:
            return "HIPFFT_INVALID_SIZE";

        case HIPFFT_UNALIGNED_DATA:
            return "HIPFFT_UNALIGNED_DATA";
    }

    return "<unknown>";
}

// Function prototypes
__global__ void nonlin(hipfftComplex *psi, double dt);
__global__ void lin(hipfftComplex *psi, double *k2, double dt);
__global__ void normalize(hipfftComplex *psi, int size);

int main(void)
{                                                                          
	// Allocate and initialize the arrays
    double *x = (double*)malloc(sizeof(double) * XN);
	double *h_k2 = (double*)malloc(sizeof(double) * XN);
	hipfftComplex *h_psi = (hipfftComplex*)
										malloc(sizeof(hipfftComplex)*XN);
	hipfftComplex *h_psi_0 = (hipfftComplex*)
										malloc(sizeof(hipfftComplex)*XN);
	
	// Create transform plans
    hipfftHandle plan;
    CUFFT_SAFE_CALL(hipfftPlan1d(&plan, XN, HIPFFT_C2C, 1));

    // X and Y wave numbers
	double dkx = 2*M_PI/XN/DX;
	double *kx = (double*)malloc(XN * sizeof(double));
	for(int i = XN/2; i >= 0; i--) 
		kx[XN/2 - i]=(XN/2 - i) * dkx;
	for(int i = XN/2+1; i < XN; i++) 
		kx[i]=(i - XN) * dkx; 

	// initialize x.
	for(int i = 0; i < XN ; i++)
		x[i] = (i-XN/2)*DX;
	
	// Initial Conditions and square of wave number
	for(int i = 0; i < XN; i++)
		{
			h_psi[i].x = sqrt(2)/cosh(x[i]);
			//h_psi[i].x = 2*exp(-(x[i]*x[i]/2.0/2.0));
			h_psi[i].y = 0;
			h_psi_0[i].x = h_psi[i].x;
			h_psi_0[i].y = h_psi[i].y;
			h_k2[i] = kx[i]*kx[i];
		}   
	
	// Allocate and copy device memory
    hipfftComplex *d_psi; double *d_k2;
	hipMalloc((void **)&d_psi, sizeof(hipfftComplex)*XN);
	hipMalloc((void **)&d_k2, sizeof(double)*XN);
    hipMemcpy(d_psi, h_psi, sizeof(hipfftComplex)*XN, hipMemcpyHostToDevice);
    hipMemcpy(d_k2, h_k2, sizeof(double)*XN, hipMemcpyHostToDevice);
	
	// initialize the grid
	dim3 threadsPerBlock(128,1,1);
	dim3 blocksPerGrid((XN + 127)/128,1,1);

	for (int i = 1; i < TN; i++)
	{
		// forward transform
    	CUFFT_SAFE_CALL(hipfftExecC2C(plan, d_psi, d_psi, HIPFFT_FORWARD));
		// linear calculation
		lin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, d_k2, DT/2);  
		// backward transform
    	CUFFT_SAFE_CALL(hipfftExecC2C(plan, d_psi, d_psi, HIPFFT_BACKWARD));
		// normalize the transform
		normalize<<<blocksPerGrid, threadsPerBlock>>>(d_psi, XN);
		// nonlinear calculation
		nonlin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, DT);
		// forward transform
    	CUFFT_SAFE_CALL(hipfftExecC2C(plan, d_psi, d_psi, HIPFFT_FORWARD));
		// linear calculation
		lin<<<blocksPerGrid, threadsPerBlock>>>(d_psi, d_k2, DT/2);  
		// backward transform
    	CUFFT_SAFE_CALL(hipfftExecC2C(plan, d_psi, d_psi, HIPFFT_BACKWARD));
		// normalize the transform
		normalize<<<blocksPerGrid, threadsPerBlock>>>(d_psi, XN);
	}

	hipMemcpy(h_psi, d_psi, sizeof(hipfftComplex)*XN, hipMemcpyDeviceToHost);
	// plot results
	cm_plot_1df(h_psi_0, h_psi, LX, XN, "plottingf.m");

	// garbage collection
	hipfftDestroy(plan);
	free(x);
	free(h_k2);
	free(kx);
    free(h_psi_0);
	free(h_psi);
	hipFree(d_psi);
	hipFree(d_k2);
	return 0;
}

__global__ void nonlin(hipfftComplex *psi, double dt)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
    
	double psi2 = hipCabsf(psi[i])*hipCabsf(psi[i]);
    hipfftComplex expo = make_hipComplex(cos(psi2*dt), sin(psi2*dt));
	psi[i] = hipCmulf(psi[i], expo);
}

__global__ void lin(hipfftComplex *psi, double *k2, double dt)
{                  
	int i = threadIdx.x + blockIdx.x * blockDim.x; 
	
    hipfftComplex expo = make_hipComplex(cos(k2[i]*dt), -sin(k2[i]*dt));
	psi[i] = hipCmulf(psi[i], expo);
}

__global__ void normalize(hipfftComplex *psi, int size)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x; 

	psi[i].x = psi[i].x/size; psi[i].y = psi[i].y/size;
}
